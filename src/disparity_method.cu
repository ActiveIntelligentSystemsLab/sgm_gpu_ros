/***********************************************************************
  Copyright (C) 2019 Hironori Fujimoto

  This program is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
  You should have received a copy of the GNU General Public License
  along with this program.  If not, see <http://www.gnu.org/licenses/>.
***********************************************************************/

#include "sgm_gpu/disparity_method.h"

namespace sgm_gpu
{

static hipStream_t stream1, stream2, stream3;//, stream4, stream5, stream6, stream7, stream8;
static uint8_t *d_im0;
static uint8_t *d_im1;
static cost_t *d_transform0;
static cost_t *d_transform1;
static uint8_t *d_cost;
static uint8_t *d_disparity;
static uint8_t *d_disparity_filtered_uchar;
static uint8_t *h_disparity;
static uint8_t *d_disparity_right;
static uint8_t *d_disparity_right_filtered_uchar;
static uint8_t *d_L0;
static uint8_t *d_L1;
static uint8_t *d_L2;
static uint8_t *d_L3;
static uint8_t *d_L4;
static uint8_t *d_L5;
static uint8_t *d_L6;
static uint8_t *d_L7;
static uint16_t *d_s;
static uint8_t p1, p2;
static bool first_alloc;
static uint32_t cols, rows, size, size_cube_l;

void init_disparity_method(const uint8_t _p1, const uint8_t _p2) {
  // We are not using shared memory, use L1
  //CUDA_CHECK_RETURN(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
  //CUDA_CHECK_RETURN(hipDeviceSetCacheConfig(hipFuncCachePreferShared));

  // Create streams
  CUDA_CHECK_RETURN(hipStreamCreate(&stream1));
  CUDA_CHECK_RETURN(hipStreamCreate(&stream2));
  CUDA_CHECK_RETURN(hipStreamCreate(&stream3));
  first_alloc = true;
  p1 = _p1;
  p2 = _p2;
  rows = 0;
  cols = 0;
}

void compute_disparity_method(cv::Mat left, cv::Mat right, cv::Mat* disparity, float *elapsed_time_ms, bool check_consistency) {
  if(cols != left.cols || rows != left.rows) {
    debug_log("WARNING: cols or rows are different");
    if(!first_alloc) {
      debug_log("Freeing memory");
      free_memory();
    }
    first_alloc = false;
    cols = left.cols;
    rows = left.rows;
    size = rows*cols;
    size_cube_l = size*MAX_DISPARITY;
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_transform0, sizeof(cost_t)*size));

    CUDA_CHECK_RETURN(hipMalloc((void **)&d_transform1, sizeof(cost_t)*size));

    int size_cube = size*MAX_DISPARITY;
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_cost, sizeof(uint8_t)*size_cube));

    CUDA_CHECK_RETURN(hipMalloc((void **)&d_im0, sizeof(uint8_t)*size));

    CUDA_CHECK_RETURN(hipMalloc((void **)&d_im1, sizeof(uint8_t)*size));

    CUDA_CHECK_RETURN(hipMalloc((void **)&d_L0, sizeof(uint8_t)*size_cube_l));
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_L1, sizeof(uint8_t)*size_cube_l));
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_L2, sizeof(uint8_t)*size_cube_l));
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_L3, sizeof(uint8_t)*size_cube_l));
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_L4, sizeof(uint8_t)*size_cube_l));
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_L5, sizeof(uint8_t)*size_cube_l));
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_L6, sizeof(uint8_t)*size_cube_l));
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_L7, sizeof(uint8_t)*size_cube_l));
    
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_s, sizeof(uint16_t)*size_cube));

    CUDA_CHECK_RETURN(hipMalloc((void **)&d_disparity, sizeof(uint8_t)*size));
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_disparity_filtered_uchar, sizeof(uint8_t)*size));
    h_disparity = new uint8_t[size];
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_disparity_right, sizeof(uint8_t)*size));
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_disparity_right_filtered_uchar, sizeof(uint8_t)*size));
  }
  debug_log("Copying images to the GPU");
  CUDA_CHECK_RETURN(hipMemcpyAsync(d_im0, left.ptr<uint8_t>(), sizeof(uint8_t)*size, hipMemcpyHostToDevice, stream1));
  CUDA_CHECK_RETURN(hipMemcpyAsync(d_im1, right.ptr<uint8_t>(), sizeof(uint8_t)*size, hipMemcpyHostToDevice, stream1));

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  dim3 block_size;
  block_size.x = 32;
  block_size.y = 32;

  dim3 grid_size;
  grid_size.x = (cols+block_size.x-1) / block_size.x;
  grid_size.y = (rows+block_size.y-1) / block_size.y;

  debug_log("Calling CSCT");
  CenterSymmetricCensusKernelSM2<<<grid_size, block_size, 0, stream1>>>(d_im0, d_im1, d_transform0, d_transform1, rows, cols);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Error: %s %d\n", hipGetErrorString(err), err);
    exit(-1);
  }

  // Hamming distance
  CUDA_CHECK_RETURN(hipStreamSynchronize(stream1));
  debug_log("Calling Hamming Distance");
  HammingDistanceCostKernel<<<rows, MAX_DISPARITY, 0, stream1>>>(d_transform0, d_transform1, d_cost, rows, cols);
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Error: %s %d\n", hipGetErrorString(err), err);
    exit(-1);
  }

  // Cost Aggregation
  const int PIXELS_PER_BLOCK = COSTAGG_BLOCKSIZE/WARP_SIZE;
  const int PIXELS_PER_BLOCK_HORIZ = COSTAGG_BLOCKSIZE_HORIZ/WARP_SIZE;

  debug_log("Calling Left to Right");
  CostAggregationKernelLeftToRight<<<(rows+PIXELS_PER_BLOCK_HORIZ-1)/PIXELS_PER_BLOCK_HORIZ, COSTAGG_BLOCKSIZE_HORIZ, 0, stream2>>>(d_cost, d_L0, d_s, p1, p2, rows, cols, d_transform0, d_transform1, d_disparity, d_L0, d_L1, d_L2, d_L3, d_L4, d_L5, d_L6);
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Error: %s %d\n", hipGetErrorString(err), err);
    exit(-1);
  }
  debug_log("Calling Right to Left");
  CostAggregationKernelRightToLeft<<<(rows+PIXELS_PER_BLOCK_HORIZ-1)/PIXELS_PER_BLOCK_HORIZ, COSTAGG_BLOCKSIZE_HORIZ, 0, stream3>>>(d_cost, d_L1, d_s, p1, p2, rows, cols, d_transform0, d_transform1, d_disparity, d_L0, d_L1, d_L2, d_L3, d_L4, d_L5, d_L6);
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Error: %s %d\n", hipGetErrorString(err), err);
    exit(-1);
  }
  debug_log("Calling Up to Down");
  CostAggregationKernelUpToDown<<<(cols+PIXELS_PER_BLOCK-1)/PIXELS_PER_BLOCK, COSTAGG_BLOCKSIZE, 0, stream1>>>(d_cost, d_L2, d_s, p1, p2, rows, cols, d_transform0, d_transform1, d_disparity, d_L0, d_L1, d_L2, d_L3, d_L4, d_L5, d_L6);
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Error: %s %d\n", hipGetErrorString(err), err);
    exit(-1);
  }
  CUDA_CHECK_RETURN(hipDeviceSynchronize());
  debug_log("Calling Down to Up");
  CostAggregationKernelDownToUp<<<(cols+PIXELS_PER_BLOCK-1)/PIXELS_PER_BLOCK, COSTAGG_BLOCKSIZE, 0, stream1>>>(d_cost, d_L3, d_s, p1, p2, rows, cols, d_transform0, d_transform1, d_disparity, d_L0, d_L1, d_L2, d_L3, d_L4, d_L5, d_L6);
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Error: %s %d\n", hipGetErrorString(err), err);
    exit(-1);
  }
  CostAggregationKernelDiagonalDownUpLeftRight<<<(cols+PIXELS_PER_BLOCK-1)/PIXELS_PER_BLOCK, COSTAGG_BLOCKSIZE, 0, stream1>>>(d_cost, d_L4, d_s, p1, p2, rows, cols, d_transform0, d_transform1, d_disparity, d_L0, d_L1, d_L2, d_L3, d_L4, d_L5, d_L6);
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Error: %s %d\n", hipGetErrorString(err), err);
    exit(-1);
  }
  CostAggregationKernelDiagonalUpDownLeftRight<<<(cols+PIXELS_PER_BLOCK-1)/PIXELS_PER_BLOCK, COSTAGG_BLOCKSIZE, 0, stream1>>>(d_cost, d_L5, d_s, p1, p2, rows, cols, d_transform0, d_transform1, d_disparity, d_L0, d_L1, d_L2, d_L3, d_L4, d_L5, d_L6);
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Error: %s %d\n", hipGetErrorString(err), err);
    exit(-1);
  }

  CostAggregationKernelDiagonalDownUpRightLeft<<<(cols+PIXELS_PER_BLOCK-1)/PIXELS_PER_BLOCK, COSTAGG_BLOCKSIZE, 0, stream1>>>(d_cost, d_L6, d_s, p1, p2, rows, cols, d_transform0, d_transform1, d_disparity, d_L0, d_L1, d_L2, d_L3, d_L4, d_L5, d_L6);
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Error: %s %d\n", hipGetErrorString(err), err);
    exit(-1);
  }
  CostAggregationKernelDiagonalUpDownRightLeft<<<(cols+PIXELS_PER_BLOCK-1)/PIXELS_PER_BLOCK, COSTAGG_BLOCKSIZE, 0, stream1>>>(d_cost, d_L7, d_s, p1, p2, rows, cols, d_transform0, d_transform1, d_disparity, d_L0, d_L1, d_L2, d_L3, d_L4, d_L5, d_L6);
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Error: %s %d\n", hipGetErrorString(err), err);
    exit(-1);
  }

  debug_log("Calling Median Filter");
  MedianFilter3x3<<<(size+MAX_DISPARITY-1)/MAX_DISPARITY, MAX_DISPARITY, 0, stream1>>>(d_disparity, d_disparity_filtered_uchar, rows, cols);
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Error: %s %d\n", hipGetErrorString(err), err);
    exit(-1);
  }
  
  if (check_consistency) {
    debug_log("Choose right disparity");
    ChooseRightDisparity<<<grid_size, block_size, 0, stream1>>>(d_disparity_right, d_s, rows, cols);
    err = hipGetLastError();
    if (err != hipSuccess) {
      printf("Error: %s %d\n", hipGetErrorString(err), err);
      exit(-1);
    }
    
    MedianFilter3x3<<<(size+MAX_DISPARITY-1)/MAX_DISPARITY, MAX_DISPARITY, 0, stream1>>>(d_disparity_right, d_disparity_right_filtered_uchar, rows, cols);
    err = hipGetLastError();
    if (err != hipSuccess) {
      printf("Error: %s %d\n", hipGetErrorString(err), err);
      exit(-1);
    }
    
    debug_log("Check left-right consistency");
    LeftRightConsistencyCheck<<<grid_size, block_size, 0, stream1>>>(d_disparity_filtered_uchar, d_disparity_right_filtered_uchar, rows, cols);
    err = hipGetLastError();
    if (err != hipSuccess) {
      printf("Error: %s %d\n", hipGetErrorString(err), err);
      exit(-1);
    }
  }
  
  hipEventRecord(stop, 0);
  
  CUDA_CHECK_RETURN(hipDeviceSynchronize());
  hipEventElapsedTime(elapsed_time_ms, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  debug_log("Copying final disparity to CPU");
  CUDA_CHECK_RETURN(hipMemcpy(h_disparity, d_disparity_filtered_uchar, sizeof(uint8_t)*size, hipMemcpyDeviceToHost));

  *disparity = cv::Mat(rows, cols, CV_8UC1, h_disparity);
}

static void free_memory() {
  CUDA_CHECK_RETURN(hipFree(d_im0));
  CUDA_CHECK_RETURN(hipFree(d_im1));
  CUDA_CHECK_RETURN(hipFree(d_transform0));
  CUDA_CHECK_RETURN(hipFree(d_transform1));
  CUDA_CHECK_RETURN(hipFree(d_L0));
  CUDA_CHECK_RETURN(hipFree(d_L1));
  CUDA_CHECK_RETURN(hipFree(d_L2));
  CUDA_CHECK_RETURN(hipFree(d_L3));
  CUDA_CHECK_RETURN(hipFree(d_L4));
  CUDA_CHECK_RETURN(hipFree(d_L5));
  CUDA_CHECK_RETURN(hipFree(d_L6));
  CUDA_CHECK_RETURN(hipFree(d_L7));
  CUDA_CHECK_RETURN(hipFree(d_disparity));
  CUDA_CHECK_RETURN(hipFree(d_disparity_filtered_uchar));
  CUDA_CHECK_RETURN(hipFree(d_disparity_right));
  CUDA_CHECK_RETURN(hipFree(d_disparity_right_filtered_uchar));
  CUDA_CHECK_RETURN(hipFree(d_cost));
  CUDA_CHECK_RETURN(hipFree(d_s));

  delete[] h_disparity;
}

void finish_disparity_method() {
  if(!first_alloc) {
    free_memory();
    CUDA_CHECK_RETURN(hipStreamDestroy(stream1));
    CUDA_CHECK_RETURN(hipStreamDestroy(stream2));
    CUDA_CHECK_RETURN(hipStreamDestroy(stream3));
  }
}

} // namespace sgm_gpu

