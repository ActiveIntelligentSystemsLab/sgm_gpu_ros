#include "hip/hip_runtime.h"
/***********************************************************************
  Copyright (C) 2019 Hironori Fujimoto

  This program is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
  You should have received a copy of the GNU General Public License
  along with this program.  If not, see <http://www.gnu.org/licenses/>.
***********************************************************************/

#include "sgm_gpu/median_filter.h"

namespace sgm_gpu
{

__global__ void MedianFilter3x3(const uint8_t* __restrict__ d_input, uint8_t* __restrict__ d_out, const uint32_t rows, const uint32_t cols) {
  MedianFilter<3>(d_input, d_out, rows, cols);
}

}
