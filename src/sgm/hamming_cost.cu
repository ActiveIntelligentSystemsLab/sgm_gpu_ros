#include "hip/hip_runtime.h"
/**
    This file is part of sgm. (https://github.com/dhernandez0/sgm).

    Copyright (c) 2016 Daniel Hernandez Juarez.

    sgm is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    sgm is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with sgm.  If not, see <http://www.gnu.org/licenses/>.

**/

#include "hamming_cost.h"

//d_transform0, d_transform1, d_cost, rows, cols
__global__ void
HammingDistanceCostKernel (  const cost_t *d_transform0, const cost_t *d_transform1,
		uint8_t *d_cost, const int rows, const int cols ) {
	//const int Dmax=   blockDim.x;  // Dmax is CTA size
	const int y=      blockIdx.x;  // y is CTA Identifier
	const int THRid = threadIdx.x; // THRid is Thread Identifier

	__shared__ cost_t SharedMatch[2*MAX_DISPARITY];
	__shared__ cost_t SharedBase [MAX_DISPARITY];

	SharedMatch [MAX_DISPARITY+THRid] = d_transform1[y*cols+0];  // init position

	int n_iter = cols/MAX_DISPARITY;
	for (int ix=0; ix<n_iter; ix++) {
		const int x = ix*MAX_DISPARITY;
		SharedMatch [THRid]      = SharedMatch [THRid + MAX_DISPARITY];
		SharedMatch [THRid+MAX_DISPARITY] = d_transform1 [y*cols+x+THRid];
		SharedBase  [THRid]      = d_transform0 [y*cols+x+THRid];

		__syncthreads();
		for (int i=0; i<MAX_DISPARITY; i++) {
			const cost_t base  = SharedBase [i];
			const cost_t match = SharedMatch[(MAX_DISPARITY-1-THRid)+1+i];
			d_cost[(y*cols+x+i)*MAX_DISPARITY+THRid] = popcount( base ^ match );
		}
		__syncthreads();
	}
	// For images with cols not multiples of MAX_DISPARITY
	const int x = MAX_DISPARITY*(cols/MAX_DISPARITY);
	const int left = cols-x;
	if(left > 0) {
		SharedMatch [THRid]      = SharedMatch [THRid + MAX_DISPARITY];
		if(THRid < left) {
			SharedMatch [THRid+MAX_DISPARITY] = d_transform1 [y*cols+x+THRid];
			SharedBase  [THRid]      = d_transform0 [y*cols+x+THRid];
		}

		__syncthreads();
		for (int i=0; i<left; i++) {
			const cost_t base  = SharedBase [i];
			const cost_t match = SharedMatch[(MAX_DISPARITY-1-THRid)+1+i];
			d_cost[(y*cols+x+i)*MAX_DISPARITY+THRid] = popcount( base ^ match );
		}
		__syncthreads();
	}
}
