#include "hip/hip_runtime.h"
/***********************************************************************
  Copyright (C) 2019 Hironori Fujimoto

  This program is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.
 
  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
  You should have received a copy of the GNU General Public License
  along with this program.  If not, see <http://www.gnu.org/licenses/>.
***********************************************************************/

#include "right_disparity.h"
#include "configuration.h"

__global__ void ChooseRightDisparity(uint8_t *right_disparity, const uint16_t *smoothed_cost, const uint32_t rows, const uint32_t cols) {
  const int x = blockIdx.x*blockDim.x+threadIdx.x;
  const int y = blockIdx.y*blockDim.y+threadIdx.y;
  
  int min_cost_disparity = 0;
  uint16_t min_cost = smoothed_cost[y*cols*MAX_DISPARITY + x*MAX_DISPARITY + min_cost_disparity];
  
  // Prevent overflow of index
  /*
  int limit_disparity = MAX_DISPARITY;
  if (p_x + MAX_DISPARITY >= rows) {
    limit_disparity = p_x + MAX_DISPARITY - rows - 1;
  }
  */
  
  for (int d = 1; x + d < rows; d++) {
    uint16_t tmp_cost = smoothed_cost[y*cols*MAX_DISPARITY + (x+d)*MAX_DISPARITY + d];
    if (tmp_cost < min_cost) {
      min_cost = tmp_cost;
      min_cost_disparity = d;
    }
  }
  
  right_disparity[y*cols+x] = min_cost_disparity;
}
