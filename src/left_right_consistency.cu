#include "hip/hip_runtime.h"
/***********************************************************************
  Copyright (C) 2019 Hironori Fujimoto

  This program is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.
 
  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
  You should have received a copy of the GNU General Public License
  along with this program.  If not, see <http://www.gnu.org/licenses/>.
***********************************************************************/

#include "sgm_gpu/left_right_consistency.h"
#include "sgm_gpu/configuration.h"

namespace sgm_gpu
{

__global__ void ChooseRightDisparity(uint8_t *right_disparity, const uint16_t *smoothed_cost, const uint32_t rows, const uint32_t cols) {
  const int x = blockIdx.x*blockDim.x+threadIdx.x;
  const int y = blockIdx.y*blockDim.y+threadIdx.y;
  
  if (x >= cols || y >= rows)
    return;
  
  int min_cost_disparity = 0;
  uint16_t min_cost = smoothed_cost[(y*cols + x)*MAX_DISPARITY + min_cost_disparity];
  
  for (int d = 1; d < MAX_DISPARITY; d++) {
    if (x + d >= cols)
      break;
    uint16_t tmp_cost = smoothed_cost[(y*cols + (x+d))*MAX_DISPARITY + d];
    if (tmp_cost < min_cost) {
      min_cost = tmp_cost;
      min_cost_disparity = d;
    }
  }
  
  right_disparity[y*cols+x] = min_cost_disparity;
}

__global__ void LeftRightConsistencyCheck(uint8_t* disparity, const uint8_t* disparity_right, uint32_t rows, uint32_t cols)
{
  const int x = blockIdx.x*blockDim.x+threadIdx.x;
  const int y = blockIdx.y*blockDim.y+threadIdx.y;
  
  if (x >= cols || y >= rows)
    return;
    
  const int x_right = x - disparity[y*cols + x];
  
  if (x_right < 0) {
    disparity[y*cols + x] = 255;
    return;
  }
  
  int diff = disparity[y*cols + x] - disparity_right[y*cols + x_right];
  diff = diff < 0 ? diff * -1 : diff;
  if (diff > 1) {
    disparity[y*cols + x] = 255;
  }
}

} // namespace sgm_gpu

